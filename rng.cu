#include "hip/hip_runtime.h"
#include "random.h"
#include<stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <limits>

#define LARGEST_U_INT64 0x1p64 //=2^64
#define GPU_ID 0

inline int round_up_division(int dividend, int divisor) {
  return 1+(1-dividend)/divisor;
}

namespace rng{
  __device__ uint64_t xorshift128plus(uint64_t* sp) {
    uint64_t x = sp[0],
    y = sp[1];
    sp[0] = y;
    x^= x << 23;
    x^= y ^ (x>>17) ^ (y>>26);
    sp[1]=x;
    return x+y;
  }

  __device__ double r_unif(uint64_t* seed) {
    return ( (double) xorshift128plus(seed) )/LARGEST_U_INT64;
  }

  __device__ double pseudInv_exp(double lambda, double unif_rv){
    return -lambda*log(unif_rv);
  }
  __device__ float pseudInv_exp(float lambda, double unif_rv){
    return -lambda*logf(unif_rv);
  }

  __device__ float custom_rv(float exp_rv){
    return exp_rv * cosf(exp_rv);
  }

  template<class T>
    __device__ T composed_rv(T lambda, uint64_t* seed){
      return custom_rv(pseudInv_exp(lambda, r_unif(seed)));
    }

  template<class T>
    __device__ T composed_exp(T lambda, uint64_t* seed){
      return pseudInv_exp(lambda, r_unif(seed));
    }

  template<class T>
    __global__ void r_Exp(
        int cycles, 
        int residual_threads,
        T lambda,
        uint64_t* seeds,
        T* result
        )
    {
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      T* result_slice;
      int iter = cycles;
      if(idx < residual_threads) {
        result_slice = result + idx*(++iter);
      } else {
        result_slice = result + idx*iter + residual_threads;
      }
      uint64_t* seed_slice = seeds + 2*idx;
      uint64_t local_seed[2] = {seed_slice[0], seed_slice[1]};

      while(iter--){
        result_slice[iter]= composed_exp(lambda, local_seed);
      }

      seed_slice[0]=local_seed[0];
      seed_slice[1]=local_seed[1];
    }

  __device__ float generate_max(const int maxOfN, const float lambda,  uint64_t* seed){
    float runningMax = logf(0);
    for(int ii=0; ii<maxOfN; ii++){
      runningMax = fmaxf(composed_rv(lambda, seed), runningMax);
    }
    return runningMax;
  }

  __device__ void warp_coop_max(
    const int coopNum,
    const int thread_id,
    const int maxOfN,
    const float lambda,
    uint64_t* seed,
    float* result
  ) {
    int maxNum = maxOfN/coopNum + ((thread_id%coopNum < maxOfN%coopNum) ? 1 : 0); 
    float local_result = generate_max(maxNum, lambda, seed);

    if(thread_id%coopNum == 0){
      result[thread_id/coopNum] = local_result;
    } else{
      for(int ii=1; ii<coopNum; ii++){
        if(thread_id%coopNum == ii){
          result[thread_id/coopNum] = fmaxf(result[thread_id/coopNum], local_result);
        }
      }
    }
    
  }

  template<class T>
    __global__ void max_rv(
      const int number,
      const int threadTotal,
      const int maxOfN,
      const T lambda,
      uint64_t* seeds,
      T* result
      )
  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t* seed_slice = seeds+2*idx;
    uint64_t local_seed[2] = {seed_slice[0], seed_slice[1]};
    int cycle = number/threadTotal;
    T* result_slice = result + idx*cycle;
    while(cycle--){
      result_slice[cycle] = generate_max(maxOfN, lambda, local_seed);
    }

    int warpSize = 32;

    result_slice = result + cycle*threadTotal;
    int residual = number%threadTotal;
    for(int ii=2; ii<= warpSize; ii=ii<<1){
      if(residual>=threadTotal/ii){
        residual -= threadTotal/ii;
        warp_coop_max(ii, idx, maxOfN, lambda, local_seed, result_slice);
        result_slice += threadTotal/ii;
      }
    }

    //now residual<threadTotal/warpSize
    if(idx/warpSize /*warpIndex*/ < residual){
        warp_coop_max(warpSize, idx, maxOfN, lambda, local_seed, result_slice);
    }

    seed_slice[0] = local_seed[0];
    seed_slice[1] = local_seed[1];
  }

  template double* gpu_r_exp<double>(const int, const double);
  template float* gpu_r_exp<float>(const int, const float);

  template<class T>
    T* gpu_r_exp(const int number, const T lambda) {

      Config* conf = new Config();
      int threads_per_block = conf->threadsPerBlock(GPU_ID); 
      int blocks = conf->blocks(GPU_ID);
      int par_threads =conf->totalNumThreads(GPU_ID); 

      assert(rngState->size()>=2*par_threads);


      int cycles = number/par_threads;
      int residual = number%par_threads;

      T *gpu_result; 
      int result_bytes = number * sizeof(T);
      hipMalloc((void**) &gpu_result, result_bytes);   

      uint64_t* rng_state_ptr = rngState->borrow();
      r_Exp <<<blocks, threads_per_block>>> (cycles, residual, lambda, rng_state_ptr, gpu_result);
      rngState->unborrow(rng_state_ptr);

      T* result = (T*) malloc(result_bytes);
      hipMemcpy(result, gpu_result, result_bytes, hipMemcpyDeviceToHost);
      hipFree(gpu_result);
      return result;
    }
  
  //template double* gpu_max_rv<double>(const int, const int, const double);
  template float* gpu_max_rv<float>(const int, const int, const float);

  template<class T>
    T* gpu_max_rv(const int number, const int maxOfN, const T lambda){
      Config* conf = new Config();
      int threads_per_block = conf->threadsPerBlock(GPU_ID);
      int blocks = conf->blocks(GPU_ID);
      int threads = conf->totalNumThreads(GPU_ID);
      
      assert(rngState->size()>=2*threads);

      
      T *gpu_result;
      int result_bytes = number * sizeof(T);
      hipMalloc((void**) &gpu_result, result_bytes);

      uint64_t* rng_state_ptr = rngState->borrow();
      max_rv <<<blocks, threads_per_block>>> (number, threads, maxOfN, lambda, rng_state_ptr, gpu_result);
      rngState->unborrow(rng_state_ptr);

      T* result = (T*) malloc(result_bytes);
      hipMemcpy(result, gpu_result, result_bytes, hipMemcpyDeviceToHost);
      hipFree(gpu_result);
      return result;
    }
}
