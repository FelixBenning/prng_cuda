#include "hip/hip_runtime.h"
#include "random.h"
#include<stdlib.h>
#include <stdio.h>
#include <assert.h>

#define LARGEST_U_INT64 0x1p64 //=2^64
#define GPU_ID 0

inline int round_up_division(int dividend, int divisor) {
  return 1+(1-dividend)/divisor;
}

namespace rng{
  __device__ uint64_t xorshift128plus(uint64_t* sp) {
    uint64_t x = sp[0],
    y = sp[1];
    sp[0] = y;
    x^= x << 23;
    x^= y ^ (x>>17) ^ (y>>26);
    sp[1]=x;
    return x+y;
  }

  __device__ double r_unif(uint64_t* seed) {
    return ( (double) xorshift128plus(seed) )/LARGEST_U_INT64;
  }

  __device__ double pseudInv_exp(double lambda, double unif_rv){
    return -lambda*log(unif_rv);
  }

  __device__ float pseudInv_exp(float lambda, double unif_rv){
    return -lambda*logf(unif_rv);
  }

  __device__ float custom_rv(float exp_rv){
    return exp_rv * cosf(exp_rv);
  }

  template<class T>
    __device__ T composed_custom(T lambda, uint64_t* seed){
      return custom_rv(pseudoInv_exp(lambda, r_unif(seed)));
    }

  template<class T>
    __device__ T composed_exp(T lambda, uint64_t* seed){
      return pseudInv_exp(lambda, r_unif(seed));
    }

  template<class T>
    __global__ void r_Exp(
        int cycles, 
        int residual_threads,
        T lambda,
        uint64_t* seeds,
        T* result
        )
    {
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      T* result_slice;
      int iter = cycles;
      if(idx < residual_threads) {
        result_slice = result + idx*(++iter);
      } else {
        result_slice = result + idx*iter + residual_threads;
      }
      uint64_t* seed_slice = seeds + 2*idx;
      uint64_t local_seed[2] = {seed_slice[0], seed_slice[1]};

      while(iter--){
        result_slice[iter]= composed_exp(lambda, local_seed);
      }

      seed_slice[0]=local_seed[0];
      seed_slice[1]=local_seed[1];
    }

  template double* gpu_r_exp<double>(const int, const double);
  template float* gpu_r_exp<float>(const int, const float);

  template<class T>
    T* gpu_r_exp(const int number, const T lambda) {

      Config* conf = new Config();
      int threads_per_block = conf->threadsPerBlock(GPU_ID); 
      int blocks = conf->blocks(GPU_ID);
      int par_threads =conf->totalNumThreads(GPU_ID); 

      assert(rngState->size()>=2*par_threads);


      int cycles = number/par_threads;
      int residual_threads = number%par_threads;

      T *gpu_result; 
      int result_bytes = number * sizeof(T);
      hipMalloc((void**) &gpu_result, result_bytes);   

      uint64_t* rng_state_ptr = rngState->borrow();
      r_Exp <<<blocks, threads_per_block>>> (cycles, residual_threads, lambda, rng_state_ptr, gpu_result);
      rngState->unborrow(rng_state_ptr);

      T* result = (T*) malloc(result_bytes);
      hipMemcpy(result, gpu_result, result_bytes, hipMemcpyDeviceToHost);
      hipFree(gpu_result);
      return result;
    }
}
